#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <vector>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel()
{}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
  hipSetDevice(MYDEVICE);

  // pointer for device memory
  int* d_a;

  // define grid and block size
  int numBlocks          = 8;
  int numThreadsPerBlock = 8;

  // host vector
  // hint: you might want to reserve some memory
  std::vector<int> h_a;

  // Part 1 of 5: allocate host and device memory
  size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
  hipMalloc();

  // Part 2 of 5: configure and launch kernel
  dim3 dimGrid();
  dim3 dimBlock();
  myFirstKernel<<<, >>>();

  // block until the device has completed
  hipDeviceSynchronize();

  // check if kernel execution generated an error
  checkCUDAError("kernel execution");

  // Part 4 of 5: device to host copy
  hipMemcpy();

  // Check for any CUDA errors
  checkCUDAError("hipMemcpy");

  // Part 5 of 5: verify the data returned to the host is correct
  for (int i = 0; i < 8; ++i) {
    for (int j = 0; j < 8; ++j) {
      // assert(h_a[i * numThreadsPerBlock + j] == i + j + 42);
    }
  }

  // free device memory
  hipFree(d_a);

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}

void checkCUDAError(const char* msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err)
              << std::endl;
    exit(-1);
  }
}
