#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0
__global__ void saxpy(unsigned int n, double a, double* x, double* y)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main(void)
{
  hipSetDevice(MYDEVICE);

  // 1<<N is the equivalent to 2^N
  unsigned int N = 20 * (1 << 20);
  double *x, *y, *d_x, *d_y;
  std::vector<double> x(N, 1.);
  std::vector<double> y(N, 2.);

  hipMalloc(&d_x, N * sizeof(double));
  hipMalloc(&d_y, N * sizeof(double));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x.data(), N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y.data(), N * sizeof(double), hipMemcpyHostToDevice);

  hipEventRecord(start);

  saxpy<<<(N + 511) / 512, 512>>>(N, 2.0, d_x, d_y);

  hipEventRecord(stop);

  hipMemcpy(y.data(), d_y, N * sizeof(double), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  double maxError = 0.;
  for (unsigned int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i] - 4.0));
  }

  hipFree(d_x);
  hipFree(d_y);
}
